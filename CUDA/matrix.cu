
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(int *a, int *b, int *c)
{
	int indexX = blockIdx.x * blockDim.x + threadIdx.x;
	int indexY = blockIdx.y * blockDim.y + threadIdx.y;
	c[indexX + indexY] = a[indexX + indexY] * b[indexX + indexY];
}



#define N 20
#define THREADS_PER_BLOCK 512

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc( (void **) &d_a, N * size );
	hipMalloc( (void **) &d_b, N * size );
	hipMalloc( (void **) &d_c, N * size );

	a = (int *)malloc(N * size);
	b = (int *)malloc(N * size);
	c = (int *)malloc(N * size);

	for( int i = 0; i < N * N; i++)
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	hipMemcpy( d_a, a, N * size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, N * size, hipMemcpyHostToDevice );
	
	dim3 blocos = dim3((N + (THREADS_PER_BLOCK -1)) / THREADS_PER_BLOCK, (N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, 1);
	dim3 t = dim3(THREADS_PER_BLOCK);

	vector_add<<< blocos,t >>>(d_a, d_b, d_c);

	hipMemcpy( c, d_c, N * size, hipMemcpyDeviceToHost );

	printf("c[0] = %d\n",c[0]);
	printf("c[%d] = %d\n", N*N-1, c[N*N-1]);

	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	return 0;
}

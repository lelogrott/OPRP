
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(int *a, int *b, int *c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for( int i = 0; i < N; i++)
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );
	
	vector_add<<< (N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b, d_c);

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	printf("c[0] = %d\n",c[0]);
	printf("c[%d] = %d\n", N-1, c[N-1]);

	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32
#define THREADS_PER_BLOCK 16*16
#define TILE_DIM 16

__global__ void vector_add(int* A, int* B, int* C) {

    int valor_c = 0;

    int linha = blockIdx.y * TILE_DIM + threadIdx.y;
    int coluna = blockIdx.x * TILE_DIM + threadIdx.x;
	//printf(">> %d %d\n", linha, coluna);
    __shared__ int As[TILE_DIM][TILE_DIM];
    __shared__ int Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (N*N)/(TILE_DIM*TILE_DIM); k++)
    {
     
       	As[threadIdx.y][threadIdx.x] = A[linha*N + coluna];
       	Bs[threadIdx.y][threadIdx.x] = B[linha*N + coluna];

        __syncthreads();

        for (int i = 0; i < TILE_DIM; ++i) 
        	valor_c += As[i][threadIdx.x] * Bs[threadIdx.y][i];

        __syncthreads();
    }

    if (linha < N && coluna < N)
    { 
/*    	C[linha*N + coluna] = valor_c;
    	if(linha*N + coluna == 1023 || linha*N + coluna == 0)
    		printf("valor = %d\nposicao = %d\n", valor_c, linha*N + coluna);
*/
	   	C[1023] = 500;

    }

}

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc( (void **) &d_a, N * size );
	hipMalloc( (void **) &d_b, N * size );
	hipMalloc( (void **) &d_c, N * size );

	a = (int *)malloc(N * size);
	b = (int *)malloc(N * size);
	c = (int *)malloc(N * size);

	for( int i = 0; i < N * N; i++)
	{
		a[i] = b[i] = 1;
		c[i] = 0;
	}
	
	
	hipMemcpy( d_a, a, N * size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, N * size, hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, N * size, hipMemcpyHostToDevice );
	
	dim3 blocos = dim3((N*N)/THREADS_PER_BLOCK, (N*N)/THREADS_PER_BLOCK, 1);
	dim3 t = dim3(16,16);

	vector_add<<< blocos,t >>>(d_a, d_b, d_c);

	hipMemcpy( c, d_c, N * size, hipMemcpyDeviceToHost );

	printf("c[0] = %d\n",c[0]);
	printf("c[%d] = %d\n", N*N-1, c[N*N-1]);
	printf("size = %d\n",size);
	
	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	hipDeviceReset();

	return 0;
}

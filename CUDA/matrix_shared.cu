#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 3
#define THREADS_PER_BLOCK 512


__global__ void vector_add(int *a, int *b, int *c)
{
	int indexX = blockIdx.x * blockDim.x + threadIdx.x;
	int indexY = blockIdx.y * blockDim.y + threadIdx.y;
	int comeco,j = (indexX + indexY) - ((indexX + indexY)/N) * N;
	comeco = (indexX + indexY) - (indexX + indexY)%N; 
	for(int i = 0; i < N; i++)
	{
		c[indexX + indexY] += a[comeco + i] * b[j];
		j+=N;
	}
	//printf("\n\n>> threadID = %d; X = %d; Y = %d\n\n", threadIdx.x, indexX, indexY);
	
}
__global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    float CValue = 0;

    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

         if (k*TILE_DIM + threadIdx.x < ACols && Row < ARows)   As[threadIdx.y][threadIdx.x] = A[Row*ACols + k*TILE_DIM + threadIdx.x];
         else                                                   As[threadIdx.y][threadIdx.x] = 0.0;

         if (k*TILE_DIM + threadIdx.y < BRows && Col < BCols)   Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
         else                                                   Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_DIM; ++n) CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
    }

    if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc( (void **) &d_a, N * size );
	hipMalloc( (void **) &d_b, N * size );
	hipMalloc( (void **) &d_c, N * size );

	a = (int *)malloc(N * size);
	b = (int *)malloc(N * size);
	c = (int *)malloc(N * size);

	for( int i = 0; i < N * N; i++)
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}
	
	
	hipMemcpy( d_a, a, N * size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, N * size, hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, N * size, hipMemcpyHostToDevice );
	
	dim3 blocos = dim3((N + (THREADS_PER_BLOCK -1)) / THREADS_PER_BLOCK, (N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, 1);
	dim3 t = dim3(THREADS_PER_BLOCK);

	vector_add<<< blocos,t >>>(d_a, d_b, d_c);

	hipMemcpy( c, d_c, N * size, hipMemcpyDeviceToHost );

	printf("c[0] = %d\n",c[0]);
	printf("c[%d] = %d\n", N*N-1, c[N*N-1]);

	free(a);
	free(b);
	free(c);

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );

	hipDeviceReset();

	return 0;
}
